#include "hip/hip_runtime.h"
#include <iostream>

#include "cuda_funs.h"


/*
		assign k1_32bit         = 32'h0000222C ;         // DistortCoefArray[0]  =  0.133495    ; 
		assign k2_32bit         = 32'hFFFF45F3 ;         // DistortCoefArray[1]  =  -0.726764   ; 
		assign p1_32bit         = 32'h00000000 ;         // DistortCoefArray[2]  =  0.000000    ; 
		assign p2_32bit         = 32'h00000000 ;         // DistortCoefArray[3]  =  0.000000    ; 
		assign k3_32bit         = 32'h00016D18 ;         // DistortCoefArray[4]  =  1.426151    ; 
		assign k4_32bit         = 32'h00000000 ;         // DistortCoefArray[5]  =  0.000000    ; 
		assign k5_32bit         = 32'h00000000 ;         // DistortCoefArray[6]  =  0.000000    ; 
		assign k6_32bit         = 32'h00000000 ;         // DistortCoefArray[7]  =  0.000000    ; 
    
		assign ir_32bit[0] 	    = 32'h0000006D ;    	 // iRMatrix[0]  =  0.001665   ;  
		assign ir_32bit[1] 	    = 32'h00000000 ;    	 // iRMatrix[1]  =  0.000004   ;  
		assign ir_32bit[2] 	    = 32'hFFFF7A5D ;    	 // iRMatrix[2]  =  -0.522026  ;  
		assign ir_32bit[3] 	    = 32'h00000000 ;    	 // iRMatrix[3]  =  -0.000004  ;  
		assign ir_32bit[4] 	    = 32'h0000006D ;    	 // iRMatrix[4]  =  0.001665   ;  
		assign ir_32bit[5] 	    = 32'hFFFFB958 ;    	 // iRMatrix[5]  =  -0.276016  ;  
		assign ir_32bit[6] 	    = 32'hFFFFFFFE ;    	 // iRMatrix[6]  =  -0.000041  ;  
		assign ir_32bit[7] 	    = 32'h00000000 ;    	 // iRMatrix[7]  =  0.000004   ;  
		assign ir_32bit[8] 	    = 32'h00010332 ;    	 // iRMatrix[8]  =  1.012488   ;

		assign u0_32bit    	    = 32'h0133F0E2 ;         // AcameraMatrix[0]  =  307.940948   ;     
		assign v0_32bit    	    = 32'h00A33E4F ;         // AcameraMatrix[1]  =  163.243393   ;     
		assign fx_32bit    	    = 32'h025877D8 ;         // AcameraMatrix[2]  =  600.468140   ;     
		assign fy_32bit    	    = 32'h025877D8 ;         // AcameraMatrix[3]  =  600.468140   ; 

*/

/*
		32*20 X 6*80
	  _______________________________________________________________
   |             |                                                 |
   |   thread    |                                                 |
   |   block     |                                                 |
   |             |                                                 |
   |_____________|                                                 |
   |                                                               |
   |                                                               |
   |                                                               |
   |                                                               |
   |                                                               |
   |                                                               |
   |                                                               |
   |                                                               |
   |                                                               |
   |                                                               |
   |                                                               |
   |                                                               |
   |_______________________________________________________________|

   1) each thread take cares of one destnation pixel (x,y)
   2) compute (u,v) from (x,y)
   3) texture fetch according to (u,v)
   4) colasely saving back to DDR2 memory
*/


namespace {

	static const int threadBlock_x = 32;    // TK1 has 1 SM * 192 cores/SM = 192 Cuda cores 
	static const int threadBlock_y = 6;     // only can process 1 block at a time 

	// Texture reference for 2D float texture
	static texture<float, 2, hipReadModeElementType> tex;

	__global__
		// void census_kernel(float* d_dest, int width, int height, const float* DistortCoefArray, const float* iRMatrix , const float* AcameraMatrix )
	void census_kernel(float* d_dest, int width, int height ) 
		{
             int x_dest = threadIdx.x + blockIdx.x * blockDim.x;     // col index in image
             int y_dest = threadIdx.y + blockIdx.y * blockDim.y;    //  row  index

        	/*
				x_middle_p = ir[0] * x_dest + ir[1] * y_dest + ir[2] * w_dest ;   
				y_middle_p = ir[3] * x_dest + ir[4] * y_dest + ir[5] * w_dest ;
				w_middle_p = ir[6] * x_dest + ir[7] * y_dest + ir[8] * w_dest ;

				x_middle  = x_middle_p / w_middle_p ;    // one pixel clok 
				y_middle  = y_middle_p / w_middle_p ;

			    double x2 = x*x, y2 = y*y;
			    double r2 = x2 + y2,
			    		 _2xy = 2*x*y;
			    double kr = 1 + ((k3*r2 + k2)*r2 + k1)*r2 
			    double u = fx*(x*kr + p1*_2xy + p2*(r2 + 2*xx)) + u0;
			    double v = fy*(y*kr + p1*(r2 + 2*yy) + p2*_2xy) + v0;
			*/

			float iRMatrix[9] ;
			float DistortCoefArray[8];
			float AcameraMatrix[4] ;         

            DistortCoefArray[0]  =  -0.512378  ;
            DistortCoefArray[1]  =  0.390507   ;
            DistortCoefArray[2]  =  0.000000   ;
            DistortCoefArray[3]  =  0.000000   ;
            DistortCoefArray[4]  =  -0.572942  ;
            DistortCoefArray[5]  =  0.000000   ;
            DistortCoefArray[6]  =  0.000000   ;
            DistortCoefArray[7]  =  0.000000   ;

            AcameraMatrix[0]  =  300.947083    ;
            AcameraMatrix[1]  =  177.569168    ;
            AcameraMatrix[2]  =  982.246826    ;
            AcameraMatrix[3]  =  982.246826    ;

            iRMatrix[0]       =  0.001096           ;
            iRMatrix[1]       =  -0.000012          ;
            iRMatrix[2]       =  -0.320153          ;
            iRMatrix[3]       =  0.000012           ;
            iRMatrix[4]       =  0.001097           ;
            iRMatrix[5]       =  -0.201621          ;
            iRMatrix[6]       =  0.000039           ;
            iRMatrix[7]       =  0.000002           ;
            iRMatrix[8]       =  0.988872           ;

            float x_middle_p = iRMatrix[0] * (float)x_dest + iRMatrix[1] * (float)y_dest + iRMatrix[2] ;
            float y_middle_p = iRMatrix[3] * (float)x_dest + iRMatrix[4] * (float)y_dest + iRMatrix[5] ;
            float w_middle_p = iRMatrix[6] * (float)x_dest + iRMatrix[7] * (float)y_dest + iRMatrix[8] ;
            float x_middle   = x_middle_p / w_middle_p ;
            float y_middle   = y_middle_p / w_middle_p ;

            float x2 = x_middle*x_middle ;
            float y2 = y_middle*y_middle ;
            float r2 = x2 + y2 ;
            float _2xy = 2*x_middle*y_middle ;

            float k1 = DistortCoefArray[0]  ;
            float k2 = DistortCoefArray[1]  ;
            float p1 = DistortCoefArray[2]  ;
            float p2 = DistortCoefArray[3]  ;
            float k3 = DistortCoefArray[4]  ;
            float k4 = DistortCoefArray[5]  ;
            float k5 = DistortCoefArray[6]  ;
            float k6 = DistortCoefArray[7]  ;

            float u0 = AcameraMatrix[0] ;
            float v0 = AcameraMatrix[1] ;
            float fx = AcameraMatrix[2] ;
            float fy = AcameraMatrix[3] ;

            float kr = (1 + ((k3*r2 + k2)*r2 + k1)*r2)/(1 + ((k6*r2 + k5)*r2 + k4)*r2);
            float u  = fx*(x_middle*kr + p1*_2xy + p2*(r2 + 2*x2))  + u0;
            float v  = fy*(y_middle*kr + p1*(r2 + 2*y2) + p2*_2xy)  + v0;

            u = (u)/((float)width );
            v = (v)/((float)height) ;
            // make sure u v is not greater than 1.0f

//            __syncthreads();

            d_dest[y_dest*width + x_dest] = tex2D(tex, u, v);

           //  float u =((float) (x_dest))/((float)width) * 1.3f;
           //  float v =((float) (y_dest))/((float)height) *1.3f;

           // d_dest[y_dest*width + x_dest] =  tex2D(tex, u , v);

//           if(x_dest < 15 && (y_dest < 10)){
//               printf( "index %d   =  %f  u = %f   ,  v = %f \n" , x_dest, tex2D(tex, u , v),u,v);
//           }

		}

}


namespace cuda_calls {

		void rectify(const uint8_t* h_src, const void* d_unrect_img, float* d_rect_img,int width, int height,
					 const float* DistortCoefArray, const float* iRMatrix , const float* AcameraMatrix ) 
		{

			const dim3   dimGrid((width + threadBlock_x - 1) / threadBlock_x, (height + threadBlock_y - 1) / threadBlock_y);
			const dim3   dimBlock(threadBlock_x, threadBlock_y);

			unsigned int size = width * height * sizeof(float);

            printf("width = %d   ;  height = %d \n", width, height ) ;

            float* hData  =  (float*)malloc( size);
//            CudaHelper( hipHostMalloc ( (void**)&hData, size ) ) ;   why not working ?????

			// convert gray scale image to float type 
			for(int r=0; r< height; r++)
				for(int c=0; c<width; c++)
				{
//                    temp =  temp + 1 ;
//                    if(temp > 255)
//                    {
//                        temp = 0 ;
//                    }
                    hData[ r*width + c ]  =  (float) h_src[ r*width + c ] ;
				}

//            for(int r=0; r< 15; r++)
//                for(int c=0; c<10; c++)
//                {
//                    printf(" hData value = %f \n ", hData[ r*width + c ] ) ;
//                }

		    // Allocate array and copy image data
		    hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
		    hipArray *cuArray;
		    CudaHelper(hipMallocArray(&cuArray,
		                                    &channelDesc,
		                                    width,
		                                    height));
		    CudaHelper(hipMemcpyToArray(cuArray,
                                              0,
		                                      0,
		                                      hData,
		                                      size,
		                                      hipMemcpyHostToDevice));  // copy data from host to device 

			// Set texture parameters
//		    tex.addressMode[0] = hipAddressModeWrap;
//		    tex.addressMode[1] = hipAddressModeWrap;
            tex.addressMode[0] = hipAddressModeBorder;
            tex.addressMode[1] = hipAddressModeBorder;
		    tex.filterMode = hipFilterModeLinear;
		    tex.normalized = true;    // access with normalized texture coordinates

            // Bind the array to the texture
            CudaHelper(hipBindTextureToArray(tex, cuArray, channelDesc));

            // census_kernel<<<dimGrid, dimBlock, 0 >>> (d_rect_img, width, height,  DistortCoefArray,  iRMatrix , AcameraMatrix ) ;  wrong invokation
			census_kernel<<<dimGrid, dimBlock, 0 >>> (d_rect_img, width, height ) ;

           delete [] hData ;

		}

	}
